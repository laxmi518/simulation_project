#include "hip/hip_runtime.h"
#include <stdio.h>
#include "DataType.h"
#include "DeviceProcess.h"
#include "DeviceProcess_Kernel.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/detail/type_traits.h>
#include "IO.h"
#include "Configuration.h"

#define USECDUASTREAM

//Round a / b to nearest higher integer value	
unsigned int iDivUp(unsigned int a, unsigned int b){
	return (a % b != 0) ? (a / b + 1) : (a / b);	
}
class CThreadScaler
{
private:
	Integer Dg;
	Integer Db;
public:
	CThreadScaler(Integer NumThreads)
	{
		Db = min ( BLOCK_MAX_DIM, NumThreads);
		if(Db > 0)
		{
			Dg = iDivUp(NumThreads, Db);
		}else
		{
			Dg = 0;
		}
	}
	Integer Grids()
	{
		return Dg;
	}
	Integer Blocks()
	{
		return Db;
	}
};
extern "C"
{
	/*inline void check_cuda_errors(const char *filename, const int line_number)
	{
		#ifdef DEBUG
		  hipDeviceSynchronize();
		  hipError_t error = hipGetLastError();
		  if(error != hipSuccess)
		  {
			printf("CUDA error at %s:%i: %s\n", filename, line_number, hipGetErrorString(error));
			exit(-1);
		  }
		#endif
	}*/
	CCTStatusType CalcTurbulenceViscosity(hipStream_t &Stream,Integer ComputeParticleNum)
	{
		if(ComputeParticleNum > 0)
		{
			CCTStatusType StatusType;
			CThreadScaler TS(ComputeParticleNum);
			CalcTurbulenceViscosity_Kernel<<<TS.Grids(), TS.Blocks() ,0,Stream>>>(ComputeParticleNum);
			//std::string kernelName = "CalcTurbulenceViscosity";
			//WriteConstant(kernelName,ComputeParticleNum);
			//check_cuda_errors(__FILE__, __LINE__);
			StatusType = CudaSafeCall(hipGetLastError());
			CCT_ERROR_CHECK(StatusType);
		}
		return CCT_NOERR;
	}
	//Check Particle outside compute zone starts
	CCTStatusType CheckParticleOutsideComputeZone(hipStream_t &Stream,Integer *ParticleNum)
	{
		if((*ParticleNum) > 0)
		{
			CThreadScaler TS(*ParticleNum);
			CheckParticleOutsideComputeZone_Kernel<<<TS.Grids(),TS.Blocks(),0,Stream>>>(ParticleNum);
			//check_cuda_errors(__FILE__, __LINE__);
			CCTStatusType StatusType;
			StatusType = CudaSafeCall(hipGetLastError());
			CCT_ERROR_CHECK(StatusType);
		}
		return CCT_NOERR;
	}
	//Check particle Outside Compute Zone Ends

	CCTStatusType CalcExplicitly(hipStream_t &Stream,Integer ComputeParticleNum)
	{
		if((ComputeParticleNum) > 0)
		{
			CCTStatusType StatusType;
			CThreadScaler TS(ComputeParticleNum);
			CalcExplicitly_Kernel<<<TS.Grids(),TS.Blocks(),0,Stream>>>(ComputeParticleNum);
			//check_cuda_errors(__FILE__, __LINE__);
			StatusType = CudaSafeCall(hipGetLastError());
			CCT_ERROR_CHECK(StatusType);
		}
		return CCT_NOERR;
	}
	CCTStatusType ResetTriangleTopology(hipStream_t &Stream,Integer CellNum, CCell* aCell)
	{
		if(CellNum > 0)
		{
			CThreadScaler TS(CellNum);
			ResetTriangleTopology_Kernel<<<TS.Grids(),TS.Blocks(),0,Stream>>>(CellNum, aCell);
			//check_cuda_errors(__FILE__, __LINE__);
			CCTStatusType StatusType;
			StatusType = CudaSafeCall(hipGetLastError());
			CCT_ERROR_CHECK(StatusType);
		}
		return CCT_NOERR;
	}
	CCTStatusType RegisterTriangleTopology(hipStream_t &Stream,CTriangle* daTriangle, Integer TriangleNum, CCell* daCell, Integer CellNum)
	{
		unsigned int DbTriangle = min ( BLOCK_MAX_DIM, TriangleNum);
		unsigned int DgTriangle = iDivUp(TriangleNum, DbTriangle);
		RegisterTriangleTopology_Kernel<<<DgTriangle, DbTriangle,0,Stream>>>(daTriangle, TriangleNum, daCell, CellNum);
		//check_cuda_errors(__FILE__, __LINE__);
		CCTStatusType StatusType;
		StatusType = CudaSafeCall(hipGetLastError());
		CCT_ERROR_CHECK(StatusType);
		return CCT_NOERR;
	}
	CCTStatusType UpdateTrianglePosition(hipStream_t &Stream,const Integer TriangleNum, CTriangle* daTriangles)
	{
		unsigned int DbTriangle = min ( BLOCK_MAX_DIM, TriangleNum);
		unsigned int DgTriangle = iDivUp(TriangleNum, DbTriangle);
		UpdateTrianglePosition_Kernel<<<DgTriangle, DbTriangle,0,Stream>>>(TriangleNum, daTriangles);
		//check_cuda_errors(__FILE__, __LINE__);
		CCTStatusType StatusType;
		StatusType = CudaSafeCall(hipGetLastError());
		CCT_ERROR_CHECK(StatusType);
		return CCT_NOERR;
	}
	CCTStatusType RotateTrianglePosition(hipStream_t &Stream,const Integer TriangleNum,CTriangle* daTriangles, const Integer analysisStep)
	{
		unsigned int DbTriangle = min ( BLOCK_MAX_DIM, TriangleNum);
		unsigned int DgTriangle = iDivUp(TriangleNum, DbTriangle);
		RotateTrianglePosition_Kernel<<<DgTriangle, DbTriangle,0,Stream>>>(TriangleNum, daTriangles, analysisStep);
		//check_cuda_errors(__FILE__, __LINE__);
		CCTStatusType StatusType;
		StatusType = CudaSafeCall(hipGetLastError());
		CCT_ERROR_CHECK(StatusType);
		return CCT_NOERR;
	}
	CCTStatusType ResetWallPosition(hipStream_t &Stream,const Integer TriangleNum,const Integer AnalysisStep,const CTriangle* daTriangles)
	{
		unsigned int DbTriangle = min ( BLOCK_MAX_DIM, TriangleNum);
		unsigned int DgTriangle = iDivUp(TriangleNum, DbTriangle);
		ResetWallPosition_Kernel<<<DgTriangle, DbTriangle,0,Stream>>>(TriangleNum,AnalysisStep, daTriangles);
		//check_cuda_errors(__FILE__, __LINE__);
		CCTStatusType StatusType;
		StatusType = CudaSafeCall(hipGetLastError());
		CCT_ERROR_CHECK(StatusType);
		return CCT_NOERR;
	}	

	CCTStatusType CalcSTLDistance(hipStream_t &Stream, Integer ComputeParticleNum)
	{
		if(ComputeParticleNum > 0)
		{
			CThreadScaler TS(ComputeParticleNum);
			CalcSTLDistance_Kernel<<<TS.Grids(),TS.Blocks(),0,Stream>>>(ComputeParticleNum);
			//check_cuda_errors(__FILE__, __LINE__);
			CCTStatusType StatusType = CudaSafeCall(hipGetLastError());
			CCT_ERROR_CHECK(StatusType);
		}
		return CCT_NOERR;
	}
	CCTStatusType CaculateCellIDandInitializeHash(hipStream_t &Stream,Integer ParticleNum,Integer CellNum,int* dGridParticleHash, int* dGridParticleIndex, Scalar3* particlePosition)
	{
		if(ParticleNum > 0)
		{
			CThreadScaler TS(ParticleNum);
			// calculate grid hash
			calcHashD<<<TS.Grids(), TS.Blocks() ,0,Stream>>>(ParticleNum,dGridParticleHash,dGridParticleIndex,particlePosition);
			//check_cuda_errors(__FILE__, __LINE__);
			CCTStatusType StatusType = CudaSafeCall(hipGetLastError());
			CCT_ERROR_CHECK(StatusType);
		}
		return CCT_NOERR;
	}
	
	CCTStatusType reorderDataAndFindCellStart(hipStream_t& Stream,Integer numParticles,Integer numCells , int*  gridParticleHash, int*  gridParticleIndex, int*  cellStart, int*  cellEnd)
	{
		if(numParticles > 0)
		{
			CThreadScaler TS(numParticles);
			// set all cells to empty
			hipMemsetAsync(cellStart, 0xffffffff, numCells*sizeof(int),Stream);
			hipMemsetAsync(cellEnd,   0xffffffff, numCells*sizeof(int),Stream);
			int smemSize = sizeof(int)*(TS.Blocks()+1);
			reorderDataAndFindCellStartD<<< TS.Grids(), TS.Blocks(), smemSize,Stream>>>(numParticles,numCells,gridParticleHash,	gridParticleIndex,cellStart, cellEnd);
			//check_cuda_errors(__FILE__, __LINE__);
			CCTStatusType StatusType = CudaSafeCall(hipGetLastError());
			CCT_ERROR_CHECK(StatusType);
		}
		return CCT_NOERR;
	}
	CCTStatusType SortUsingThrust(Integer MaxParticleNum, Integer * daNumberHash, Integer* daNumberIndex)
	{
		CCTStatusType  Status = CCT_NOERR;
		if(MaxParticleNum > 0)
		{
			thrust::sort_by_key(thrust::device_ptr<Integer>(daNumberHash),
								thrust::device_ptr<Integer>(daNumberHash + MaxParticleNum),
								thrust::device_ptr<Integer>(daNumberIndex));
			//check_cuda_errors(__FILE__, __LINE__);
			Status = CudaSafeCall(hipGetLastError());
			CCT_ERROR_CHECK(Status);
		}
		return Status;
	}
	CCTStatusType StableSortUsingThrust(Integer MaxParticleNum, Integer * daNumberHash, Integer* daNumberIndex)
	{
		CCTStatusType  Status = CCT_NOERR;
		if(MaxParticleNum > 0)
		{
			thrust::stable_sort_by_key(thrust::device_ptr<Integer>(daNumberHash),
									   thrust::device_ptr<Integer>(daNumberHash + MaxParticleNum),
									   thrust::device_ptr<Integer>(daNumberIndex));
			//check_cuda_errors(__FILE__, __LINE__);
			Status = CudaSafeCall(hipGetLastError());
			CCT_ERROR_CHECK(Status);
		}
		return Status;
	}
	CCTStatusType CalcDragEffect(hipStream_t &Stream,Integer ComputeParticleNum)
	{
		if((ComputeParticleNum) > 0)
		{
			CThreadScaler TS(ComputeParticleNum);
			CalcDragEffect_Kernel<<<TS.Grids(),TS.Blocks(),0,Stream>>>(ComputeParticleNum);
			//check_cuda_errors(__FILE__, __LINE__);
			CCTStatusType StatusType = CudaSafeCall(hipGetLastError());
			CCT_ERROR_CHECK(StatusType);
		}
		return CCT_NOERR;
	}
	CCTStatusType CalcExplicitPressure(hipStream_t &Stream,Integer ComputeParticleNum)
	{
		if((ComputeParticleNum) > 0)
		{
			CThreadScaler TS(ComputeParticleNum);
			CalcExplicitPressure_Kernel<<<TS.Grids(),TS.Blocks(),0,Stream>>>(ComputeParticleNum);
			//check_cuda_errors(__FILE__, __LINE__);
			CCTStatusType StatusType = CudaSafeCall(hipGetLastError());
			CCT_ERROR_CHECK(StatusType);
		}
		return CCT_NOERR;
	}
	CCTStatusType CalcExplicitPressureGradient(hipStream_t &Stream,Integer ComputeParticleNum)
	{
		if((ComputeParticleNum) > 0)
		{
			CThreadScaler TS(ComputeParticleNum);
			CalcExplicitPressureGradient_Kernel<<<TS.Grids(),TS.Blocks(),0,Stream>>>(ComputeParticleNum);
			//check_cuda_errors(__FILE__, __LINE__);
			CCTStatusType StatusType = CudaSafeCall(hipGetLastError());
			CCT_ERROR_CHECK(StatusType);
		}
		return CCT_NOERR;
	}
	CCTStatusType CalcTemperatureFactor(hipStream_t &Stream,Integer ComputeParticleNum)
	{
		if((ComputeParticleNum) > 0)
		{
			CThreadScaler TS(ComputeParticleNum);
			CalcTemperatureFactor_Kernel<<<TS.Grids(),TS.Blocks(),0,Stream>>>(ComputeParticleNum);
			//check_cuda_errors(__FILE__, __LINE__);
			CCTStatusType StatusType = CudaSafeCall(hipGetLastError());
			CCT_ERROR_CHECK(StatusType);
		}
		return CCT_NOERR;
	}

	CCTStatusType InitializeDeviceMemConst(CParameter Parameter,Integer ParticleNum,CTriangle * Triangles,Integer TriangleNum,CTriangleParameters * TriangleParameters,
		Integer MaxParticleNum, CDistance * STLDistance,Integer * StlID, CCell * Cell,Integer CellNum,
		Integer * CellStart,Integer * CellEnd,Integer * GridParticleIndex, CGridBox BoundingBox)
	{
		CCTStatusType Status;
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(CONSTANT_PARAMETER), &Parameter, sizeof(CParameter)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_ParticleNum), &ParticleNum, sizeof(Integer)));
		CCT_ERROR_CHECK(Status);		
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daTriangles), &Triangles, sizeof(Triangles)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_TriangleNum), &TriangleNum, sizeof(Integer)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daTrianglesParameters), &TriangleParameters, sizeof(TriangleParameters)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_MaxParticleNum), &MaxParticleNum, sizeof(Integer)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daSTLDistance), &STLDistance, sizeof(STLDistance)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daSTLID), &StlID, sizeof(STLID)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daCell), &Cell, sizeof(Cell)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_CellNum), &CellNum, sizeof(Integer)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_cellStart), &CellStart, sizeof(CellStart)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_cellEnd), &CellEnd, sizeof(CellEnd)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_gridParticleIndex), &GridParticleIndex, sizeof(GridParticleIndex)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(CONSTANT_BOUNDINGBOX), &BoundingBox, sizeof(CGridBox)) );
		CCT_ERROR_CHECK(Status);
		//check_cuda_errors(__FILE__, __LINE__);
		return CCT_NOERR;
	}
	CCTStatusType InitializeDeviceConstOutPutParticles(Integer * OutputParticleID, Scalar3 * OutputParticlePosition, Scalar3 * OutputParticleVelocity,
		Scalar * OutputParticlePressure, Scalar * OutputParticleDensity, Scalar * OutputParticleTemperature, Scalar * OutputParticleKineticViscosity,
		Scalar * OutputParticleSolidPhaseRate, ParticleType * OutputParticleType)
	{
		CCTStatusType Status;
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daOutputParticleID),&OutputParticleID, sizeof(OutputParticleID)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daOutputParticlePosition),&OutputParticlePosition, sizeof(OutputParticlePosition)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daOutputParticleVelocity),&OutputParticleVelocity, sizeof(OutputParticleVelocity)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daOutputParticlePressure),&OutputParticlePressure, sizeof(OutputParticlePressure)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daOutputParticleDensity),&OutputParticleDensity, sizeof(OutputParticleDensity)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daOutputParticleTemperature),&OutputParticleTemperature, sizeof(OutputParticleTemperature)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daOutputParticleKineticViscosity),&OutputParticleKineticViscosity, sizeof(OutputParticleKineticViscosity)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daOutputParticleSolidPhaseRate),&OutputParticleSolidPhaseRate, sizeof(OutputParticleSolidPhaseRate)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daOutputParticleType),&OutputParticleType, sizeof(OutputParticleType)));
		CCT_ERROR_CHECK(Status);
		//check_cuda_errors(__FILE__, __LINE__);
		return CCT_NOERR;
	}
	CCTStatusType InitializeDeviceConstInputParticles(Integer * InputParticleID, Scalar3 * InputParticlePosition, Scalar3 * InputParticleVelocity,
		Scalar * InputParticlePressure, Scalar * InputParticleDensity, Scalar * InputParticleTemperature, Scalar * InputParticleKineticViscosity,
		Scalar * InputParticleSolidPhaseRate, ParticleType * InputParticleType, Scalar* ParticleTurbulaceViscosity, Scalar* ParticleStrainTensorProduct)
	{
		CCTStatusType Status;
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daParticleID),&InputParticleID, sizeof(InputParticleID)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daParticlePosition),&InputParticlePosition, sizeof(InputParticlePosition)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daParticleVelocity),&InputParticleVelocity, sizeof(InputParticleVelocity)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daParticlePressure),&InputParticlePressure, sizeof(InputParticlePressure)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daParticleDensity),&InputParticleDensity, sizeof(InputParticleDensity)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daParticleTemperature),&InputParticleTemperature, sizeof(InputParticleTemperature)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daParticleKineticViscosity),&InputParticleKineticViscosity, sizeof(InputParticleKineticViscosity)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daParticleSolidPhaseRate),&InputParticleSolidPhaseRate, sizeof(InputParticleSolidPhaseRate)));
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daParticleType),&InputParticleType, sizeof(InputParticleType)));
		CCT_ERROR_CHECK(Status);
		//Turbulace 
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daParticleTurbulaceViscosity),&ParticleTurbulaceViscosity, sizeof(ParticleTurbulaceViscosity)));		
		CCT_ERROR_CHECK(Status);
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daParticleStrainTensorProduct),&ParticleStrainTensorProduct, sizeof(ParticleStrainTensorProduct)));		
		CCT_ERROR_CHECK(Status);
		//check_cuda_errors(__FILE__, __LINE__);
		return CCT_NOERR;
	}
	CCTStatusType ParticleNumberToConst(Integer ParticleNum)
	{
		CCTStatusType Status;
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_ParticleNum), &ParticleNum, sizeof(Integer)));
		//check_cuda_errors(__FILE__, __LINE__);
		CCT_ERROR_CHECK(Status);
		return CCT_NOERR;
	}
	CCTStatusType DragParametersToConst(DragParameter *InputDragParameter,Scalar3 *InputDragAcc,Scalar* InputDragTemperature ,Integer DragTriangleNum,Integer * MagnifierCount,CDragTriangle * DragTriangles)
	{
		CCTStatusType Status;
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daSTLDragParameter),&InputDragParameter, sizeof(InputDragParameter)));
		CCT_ERROR_CHECK(Status);

		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daDragAcc),&InputDragAcc, sizeof(InputDragAcc)));
		CCT_ERROR_CHECK(Status);

		//For Drag Temperature
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daDragTemperature),&InputDragTemperature, sizeof(InputDragTemperature)));
		CCT_ERROR_CHECK(Status);

		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_DragTriangleNum), &DragTriangleNum, sizeof(DragTriangleNum)));
		CCT_ERROR_CHECK(Status);
		
		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daMagnifierCount), &MagnifierCount, sizeof(MagnifierCount)));
		CCT_ERROR_CHECK(Status);

		Status = CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_daDragTriangles), &DragTriangles, sizeof(DragTriangles)));
		CCT_ERROR_CHECK(Status);
		//check_cuda_errors(__FILE__, __LINE__);
		return CCT_NOERR;
	}

}